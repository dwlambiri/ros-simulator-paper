
#include <hip/hip_runtime.h>


extern "C" __global__  void cuda_adjustPROD(int n_axons, float* hProd, float* sProd, float toxMult)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	if (n < n_axons)
	{
		hProd[n] = hProd[n]*toxMult;
		sProd[n] = sProd[n]*toxMult;
	}
}

