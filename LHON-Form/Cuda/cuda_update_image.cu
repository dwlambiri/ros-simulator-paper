
#include <hip/hip_runtime.h>
﻿
__device__ void setColor(int showRGBSox, float f, unsigned char& red, unsigned char& green, unsigned char& blue) {
	red = (unsigned char)(f); // 0 - 255
	if (showRGBSox) {
		if (f == 0) { red = 0; green = 0; blue = 0; }
		else if (f < 32) { red = 0; green = 0; blue = f * 8; }
	    else if (f < 64) { red = 0; green = 4 * f; blue = 255; }
		else if (f < 128) { red = 0; blue = 255 + 4 * (64 - f); green = 255; }
		else if (f < 192) { red = 4 * (f - 128); blue = 0; green = 255; }
		else { green = 255 + 4 * (192 - f); blue = 0; red = 255; }
#if 0
		float gt = f - red;
		if (red < 32) {
			red = red * 4;
			green = red;
		}
		if (red == 0) {
			float g = gt * 255;
			green = (unsigned char)(g);
			red = green;
			gt = g- green;
			if (green == 0) {
				blue = (unsigned char)(gt * 255);
			}
		}
#endif
	}
}

#define AXON_DISPLAY_HEALTHY 1
#define AXON_DISPLAY_STRESS 2
#define AXON_DISPLAY_DEAD 0

#define GLIA_DISPLAY_HEALTHY 5
#define GLIA_DISPLAY_STRESS 6
#define GLIA_DISPLAY_DEAD 4

#define HEALTHY_GREEN 230
#define STRESS_GREEN 127
#define DEAD_BLUE 255


/**

			show_opts[0] = chk_show_live_axons.Checked;
			show_opts[1] = chk_show_dead_axons.Checked;
			show_opts[2] = chk_show_stress.Checked;
			show_opts[3] = chk_show_tox.Checked;
			show_opts[4] = checkBox_show_rgc.Checked;
			show_opts[5] = checkBox_show_glia.Checked;
*/

extern "C" __global__  void cuda_update_image(unsigned short im_size, 
	unsigned short bmp_im_size, 
	float bmp_image_compression_ratio, 
	float bmp_z_compression_ratio,
	unsigned char* bmp, 
	float* bmp_tox, 
	float* tox, 
	unsigned char* simulation_point_mask, 
	unsigned char* init_insult_mask, 
	float tox_max, 
	bool* show_opts, 
	int showdir, 
	int lineToDisplay, 
	int imsq, 
	int head, 
	int no3d, 
	int showRGBSox, 
	int displayAtTop, 
	unsigned int* img_axon_center, 
	unsigned int* img_glia_center,
	float prodConv)
{
	int x_bmp = blockIdx.x * blockDim.x + threadIdx.x;
	int y_bmp = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_bmp < bmp_im_size && y_bmp > 0) {

		int xy_bmp = x_bmp * bmp_im_size + y_bmp;
		int xy4_bmp = xy_bmp * 4;
		

		unsigned char red = 0, green = 0, blue = 0;
		float tox_pix_value = 0;
		
		switch (showdir) {
		case 1: { 
			// XZ (vertical slider)
			//green = blue = 0;

			if (displayAtTop) {
				if (show_opts[3]) {
					float xf = x_bmp * bmp_z_compression_ratio;
					int layer = (head+(int)(xf)) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - lineToDisplay) * bmp_image_compression_ratio);
					int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					setColor(showRGBSox, tmp * 255, red, green, blue);
				}
			}
			else {
				if ((x_bmp >= lineToDisplay) && (x_bmp < lineToDisplay + (no3d>1?no3d:1)) && show_opts[3]) {
					int layer = (head + x_bmp - lineToDisplay) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - lineToDisplay) * bmp_image_compression_ratio);
					int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					setColor(showRGBSox, tmp * 255, red, green, blue);
					//red = 255;
				}
			}
			break;
		}
		case 2: {
			// YZ (horizontal slider)
			//green = blue = 0;
			if (displayAtTop) {
				if (show_opts[3]) {
					float yf = y_bmp * bmp_z_compression_ratio;
					int layer = (head + (int)(yf)) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
					int ypos = (int)((float)lineToDisplay * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					setColor(showRGBSox, tmp * 255, red, green, blue);
				}
			}
			else {
				if ((y_bmp >= lineToDisplay) && (y_bmp < lineToDisplay + (no3d > 1 ? no3d : 1)) && show_opts[3]) {
					int layer = (head + y_bmp - lineToDisplay) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
					int ypos = (int)((float)lineToDisplay * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					setColor(showRGBSox, tmp * 255, red, green, blue);
				}
			}
			break;
		}
		default: {
			int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
			int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
			int xy = ypos * im_size + xpos;
			//int xy = xpos * im_size + ypos;
			tox_pix_value = tox[imsq * lineToDisplay + xy];
			float tmp = tox_pix_value / tox_max;
			if (tmp > 1) tmp = 1;
			
			
			if (show_opts[0]) {
				if (show_opts[4] && (simulation_point_mask[xy] == AXON_DISPLAY_HEALTHY || simulation_point_mask[img_axon_center[xy_bmp]] == AXON_DISPLAY_HEALTHY)) { green = HEALTHY_GREEN; } // rgc healthy
				if (show_opts[5] && (simulation_point_mask[xy] == GLIA_DISPLAY_HEALTHY || simulation_point_mask[img_glia_center[xy_bmp]] == GLIA_DISPLAY_HEALTHY)) { green = HEALTHY_GREEN; } // glia healthy

																																												   //if (simulation_point_mask[xy] == 2) { green = 0; } // dead
				//blue = 0;
			}
			if (show_opts[1]) {
				//if (simulation_point_mask[xy] == 1) { blue = 0; } // live
				if (show_opts[4] && (simulation_point_mask[xy] == AXON_DISPLAY_DEAD || simulation_point_mask[img_axon_center[xy_bmp]] == AXON_DISPLAY_DEAD)) { blue = DEAD_BLUE; } // rgc dead
				if (show_opts[5] && (simulation_point_mask[xy] == GLIA_DISPLAY_DEAD || simulation_point_mask[img_glia_center[xy_bmp]] == GLIA_DISPLAY_DEAD)) { blue = DEAD_BLUE; } // glia dead

																																												  //green = 0;
			}
			if (show_opts[2]) {
				//if (simulation_point_mask[xy] == 1) { blue = 0; } // live
				if (show_opts[4] && (simulation_point_mask[xy] == AXON_DISPLAY_STRESS || simulation_point_mask[img_axon_center[xy_bmp]] == AXON_DISPLAY_STRESS )) { green = STRESS_GREEN; } // stress
				if (show_opts[5] && (simulation_point_mask[xy] == GLIA_DISPLAY_STRESS || simulation_point_mask[img_glia_center[xy_bmp]] == GLIA_DISPLAY_STRESS)) { green = STRESS_GREEN; } // stress

																																												   //green = 0;
			}


			if (show_opts[3]) {
				if (show_opts[4] == 0 && show_opts[5] == 0) {
					setColor(showRGBSox, tmp * 255, red, green, blue);
				}
				else {
					red = (unsigned char)(tmp * 255); // 0 - 255
				}
				// green = 255 - normalized_toxin;
			}
			//else { red = 0; }

			if (init_insult_mask[xy_bmp]) { red = blue = green = 255; /*red = 0;*/ }
			
		}
		}
		

		bmp[xy4_bmp] = blue;
		bmp[xy4_bmp + 1] = green;
		bmp[xy4_bmp + 2] = red;

		bmp_tox[xy_bmp] = tox_pix_value*prodConv;
	}
}

/*

// Jet colormap: https://www.mathworks.com/help/matlab/ref/jet.html

if (normalized_toxin < 64) { r = 0; g = 4 * v; b = 255; }
else if (normalized_toxin < 128) { r = 0; b = 255 + 4 * (64 - v); g = 255; }
else if (normalized_toxin < 192) { r = 4 * (v - 128); b = 0; g = 255; }
else { g = 255 + 4 * (192 - normalized_toxin); b = 0; r = 255; }

*/
