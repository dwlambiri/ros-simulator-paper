
#include <hip/hip_runtime.h>
﻿#define rate_tissue_index_xy 10
#define rate_tissue_index_z 11

extern "C" __global__  void cuda_prep1(unsigned short im_size, unsigned char* pix_out_of_nerve, unsigned char* rate, int rate_dimensions)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size)
	{
		int xy = x * im_size + y;
		int xyN = xy * rate_dimensions;

		if (pix_out_of_nerve[xy]) {
			rate[xyN] = rate_tissue_index_xy;
			rate[xyN + 1] = rate_tissue_index_xy;
			rate[xyN + 2] = rate_tissue_index_xy;
			rate[xyN + 3] = rate_tissue_index_xy;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = rate_tissue_index_z;
				rate[xyN + 5] = rate_tissue_index_z;
			}
		}
		else {
			if (pix_out_of_nerve[xy + im_size]) rate[xyN] = rate_tissue_index_xy;
			if (pix_out_of_nerve[xy - im_size]) rate[xyN + 1] = rate_tissue_index_xy;
			if (pix_out_of_nerve[xy + 1])		rate[xyN + 2] = rate_tissue_index_xy;
			if (pix_out_of_nerve[xy - 1])		rate[xyN + 3] = rate_tissue_index_xy;
		}
	}
}

// Set nerve boundary rates to 0
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int[,] neighbors = new int[,] { { x + 1, y }, { x - 1, y }, { x, y + 1 }, { x, y - 1 } };
//        for (uint k = 0; k < 4; k++)
//            if (pix_out_of_nerve[x, y] || pix_out_of_nerve[neighbors[k, 0], neighbors[k, 1]])
//                rate[x, y, k] = 0;
//    }
