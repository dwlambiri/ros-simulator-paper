
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_diffusion2D(int* pix_idx, int pix_idx_num, unsigned int im_size, int index_old,
	int index_new, float* tox, float* detox, float* tox_prod, unsigned char* rate, float* rate_values)
{
	
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		int xy0 = xy + im_size;
		int xy1 = xy - im_size;
		int xy2 = xy + 1;
		int xy3 = xy - 1;

		//int xyN = xy * rate_dimensions;
		//[DWL] 2D sim has 4 neigbours. mult by 4 same as shift to left by 2
		int xyN = xy << 2;
			
	    float* tox_old = &tox[index_old];
		float* tox_new = &tox[index_new];
		
		int rn = rate[xyN];
		int rn1 = rate[xyN + 1];
		int rn2 = rate[xyN + 2];
		int rn3 = rate[xyN + 3];

		float t = tox_old[xy];

		float cnew = t +
			(tox_old[xy0] - t) * rate_values[rn] +
			(tox_old[xy1] - t) * rate_values[rn1] +
			(tox_old[xy2] - t) * rate_values[rn2] +
			(tox_old[xy3] - t) * rate_values[rn3];

		tox_new[xy] = cnew * detox[xy] + (tox_prod[xy]>=0? tox_prod[xy]:0);

		//float dex = detox[xy] * t;
		//tox_new[xy] = ((cnew < dex) ? 0 : cnew - dex);
	}
}
