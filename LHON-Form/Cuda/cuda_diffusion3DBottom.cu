
#include <hip/hip_runtime.h>
﻿
#define rateUpLayerIndex  4
#define rateDownLayerIndex  5
#define rate_values_size 13

/*
private readonly int voxelNoProd_c = -1;
        private readonly int tissueProd_c = 0;
        private readonly int voxelGliaMito_c = 1;
        private readonly int voxelGliaProd_c = 2;
        private readonly int voxelMembrane_c= 3;
        private readonly int voxelRgcMito_c = 4;
        private readonly int voxelRgcProd_c = 5;
*/

#define voxelRgcMito_c  4
#define voxelRgcProd_c  5

extern "C" __global__  void cuda_diffusion3DBottom(int* pix_idx, int pix_idx_num, unsigned short im_size,
	float* tox, float* detox, float* tox_prod, unsigned char* rate, float* rate_values_array, int layerNo,
	int dstl, int tl, int ml, int bl, float mult, float detoxmult, int* assignedPixelMap)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		//int sq = im_size * im_size;

		int xy0 = xy + im_size;
		int xy1 = xy - im_size;
		int xy2 = xy + 1;
		int xy3 = xy - 1;
		
		// [DWL] in the 3D sim there are 6 neighbours
		int xyN = xy * 6;

		float* tox_new = &tox[dstl];
	    float* tox_old = &tox[ml];
		float* tox_up = &tox[tl];
		//float* tox_down = &tox[bl];
		float* rate_values = &rate_values_array[rate_values_size * layerNo];

		float t = tox_old[xy];

		//tox_new[xy] = t +
		float cnew = t + 
			(tox_old[xy0] - t) * rate_values[rate[xyN]] +
			(tox_old[xy1] - t) * rate_values[rate[xyN + 1]] +
			(tox_old[xy2] - t) * rate_values[rate[xyN + 2]] +
			(tox_old[xy3] - t) * rate_values[rate[xyN + 3]] + 
			(tox_up[xy] - t) * rate_values[rate[xyN + rateUpLayerIndex]];

		//[DWL], below is written the way it is because we must multiply the 'K'
		//of othe scavenging differential rate equation by detoxmult
		//Since the K (by the time it is used in this file) is already passed as a value of 1-K 
		//We must convert it back to K, mulitply by detoxmult, then convert back to 1-k

		//tox_new[xy] = cnew * (1 - (1 - detox[xy]) * detoxmult) + (tox_prod[xy] >= 0 ? tox_prod[xy] : 0) * mult;
		tox_new[xy] = cnew * (1 - (1 - detox[xy]) * detoxmult) + (tox_prod[xy] >= 0 ? tox_prod[xy] : 0) * ((assignedPixelMap[xy] < 0)? 1: mult);
		//float dex = detox[xy] * t * detoxmult;
		//tox_new[xy] = (cnew < dex ? 0 : cnew - dex);
		
	}
}
