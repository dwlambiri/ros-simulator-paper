
#include <hip/hip_runtime.h>

#define diff_membrane_index 2
#define diff_membrane_dead_index 3
#define diff_dead_index 4
#define diff_dead_z_index 8
#define diff_membrane_stress_index 9

#define AXON_HEALTHY 1
#define AXON_STRESS 2
#define AXON_DEAD 0

#define AXON_DISPLAY_HEALTHY 1
#define AXON_DISPLAY_STRESS 2
#define AXON_DISPLAY_DEAD 0

#define NOTOXCONST -1

/*
* [DWL]
* Axon states:
* AXON_HEALTHY --> AXON_STRESS --> AXON_DEAD
*/

/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
		 private readonly byte rate_zero_index = 0;
        private readonly byte rate_live_index = 1;
        private readonly byte rate_membrane_index = 2;
        private readonly byte rate_membrane_dead_index = 3;
        private readonly byte rate_dead_index = 4;
        private readonly byte rate_extra_index = 5;
        private readonly byte rate_live_z_index = 6;
        private readonly byte rate_extra_z_index = 7;
        private readonly byte rate_dead_z_index = 8;
        private readonly byte rate_membrane_stress_index = 9;
        private readonly byte rate_tissue_index_xy = 10;
        private readonly byte rate_tissue_index_z = 11;
        private readonly byte rate_one_index = 12;
        private readonly byte rate_values_size = 13
*/

extern "C" __global__  void cuda_rgc_update_live(int n_axons, float* tox, unsigned char* rate, float* detox, float* tox_prod, float k_detox_extra, float* h2sThrVector_dev,
	unsigned int * axons_cent_pix, unsigned int* axons_inside_pix, unsigned int* axons_inside_pix_idx, unsigned int* axon_surr_rate, unsigned int* axon_surr_rate_idx,
	unsigned char* axon_state, unsigned char* axon_mask, int* num_alive_axons, int* num_stress_axons, int* death_itr, int iteration, int offset, int pixelNeighbourNumbers, float deathDetox, unsigned int* axon_death_timer, unsigned int iterToDeath,
	float* hProdVector_dev, float* sProdVector_dev, float* s2hThrVector_dev, float* s2dThrVector_dev, int timer_reset)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	if (n < n_axons)
	{
		// [DWL]: I made death_tox_threshold to be an array INSTEAD of a constant 
		//			This way we can set the death threshold DIFFERENTLY for each axon
		//			The death is calculated at the head of the axon
		if (axon_state[n] == AXON_STRESS) {
			if (axon_death_timer[n] > iterToDeath || tox[offset + axons_cent_pix[n]] >= s2dThrVector_dev[n]) {
				// kill the axon if either timer expires or if the threshold is larger than preset value
				int next = axons_inside_pix_idx[n + 1];
				for (int p = axons_inside_pix_idx[n]; p < next; p++)
				{
					int idx = axons_inside_pix[p];
					tox_prod[idx] = NOTOXCONST;
					detox[idx] = deathDetox;
					axon_mask[idx] = AXON_DISPLAY_DEAD; // dead axon display mask
				}
				next = axon_surr_rate_idx[n + 1];
				for (int p = axon_surr_rate_idx[n], i = 0; p < next; p++, i++) {
					unsigned int index = axon_surr_rate[p];
					if (index != 0) {
						unsigned char c = rate[index];
						if ((i % pixelNeighbourNumbers) < 4) {
							if (c == diff_membrane_index  || c == diff_membrane_stress_index)
								c = diff_membrane_dead_index;
							else
								c = diff_dead_index;
						}
						else
							c = diff_dead_z_index;
						rate[index] = c;
					}
				}

				axon_state[n] = AXON_DEAD;
			    death_itr[n] = iteration;
			    atomicAdd(&num_alive_axons[0], -1);
				atomicAdd(&num_stress_axons[0], -1);
			}
			else {
				

				if (tox[offset + axons_cent_pix[n]] <= s2hThrVector_dev[n])
				{ 	// Move axon back to healthy state
					
					if(timer_reset) axon_death_timer[n] = 0;

					int next = axons_inside_pix_idx[n + 1];
					for (int p = axons_inside_pix_idx[n]; p < next; p++)
					{
						int idx = axons_inside_pix[p];


						if (tox_prod[idx] >= 0) {
							tox_prod[idx] = hProdVector_dev[n];
						}
						
						axon_mask[idx] = AXON_DISPLAY_HEALTHY; // healthy axon display mask

					}
					next = axon_surr_rate_idx[n + 1];
#if 0
					for (int p = axon_surr_rate_idx[n], i = 0; p < next; p++, i++) {
						unsigned int index = axon_surr_rate[p];
						if (index != 0) {
							unsigned char c = rate[index];
							if ((i % pixelNeighbourNumbers) < 4) {
								if (c == diff_membrane_stress_index)
									c = diff_membrane_index;
							}
							rate[index] = c;
						}
					}
#endif					
					axon_state[n] = AXON_HEALTHY;
					atomicAdd(&num_stress_axons[0], -1);
				}
				else {
					// increment death timer
					axon_death_timer[n]++;
				}

			}
		}
		else if (axon_state[n] == AXON_HEALTHY)
		{ 	
			if (tox[offset + axons_cent_pix[n]] >= h2sThrVector_dev[n]) {
				// Move the axon to Stress state
				int next = axons_inside_pix_idx[n + 1];
				for (int p = axons_inside_pix_idx[n]; p < next; p++)
				{
					int idx = axons_inside_pix[p];

					if (tox_prod[idx] >= 0) {
						tox_prod[idx] = sProdVector_dev[n];
					}

					axon_mask[idx] = AXON_DISPLAY_STRESS; // stressed axon for display

				}
				next = axon_surr_rate_idx[n + 1];
#if 0
				for (int p = axon_surr_rate_idx[n], i = 0; p < next; p++, i++) {
					unsigned int index = axon_surr_rate[p];
					if (index != 0) {
						unsigned char c = rate[index];
						if ((i % pixelNeighbourNumbers) < 4) {
							if (c == diff_membrane_index)
								c = diff_membrane_stress_index;
						}
						rate[index] = c;
					}
				}
#endif
				axon_state[n] = AXON_STRESS;
				axon_death_timer[n]++;
				atomicAdd(&num_stress_axons[0], +1);
			}
			else {
				if(axon_death_timer[n] > 0) axon_death_timer[n]--;
			}
			
		}
	}
}

