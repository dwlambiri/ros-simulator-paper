
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_tox_sum(int* pix_idx, int pix_idx_num, float* tox, float* tox_sum, int* regionalMask, float* zone_tox_sum, int offset, int imsquare, int no3dLayers)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];
		if (no3dLayers>1) {
			float sum = 0;
			for (int i = 0; i < no3dLayers; i++) {
				sum += tox[((offset + i) % (no3dLayers + 2))*imsquare + xy];
			}
			atomicAdd(tox_sum, sum);
			if(regionalMask[xy] >= 0){
			    atomicAdd(&zone_tox_sum[regionalMask[xy]], sum);
			}
		}
		else {
			atomicAdd(tox_sum, tox[offset*imsquare + xy]);
			if(regionalMask[xy] >= 0){
				atomicAdd(&zone_tox_sum[regionalMask[xy]], (float) tox[offset*imsquare + xy]);
			}
		}
		
	}
}
