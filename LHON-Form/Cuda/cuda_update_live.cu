
#include <hip/hip_runtime.h>

#define diff_dead_index 4
#define diff_dead_z_index 8

/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
		private readonly byte rate_zero_index = 0;
		private readonly byte rate_live_index = 1;
		private readonly byte rate_membrane_index = 2;
		private readonly byte rate_UNUSED_index = 3;
		private readonly byte rate_dead_index = 4;
		private readonly byte rate_extra_index = 5;
		private readonly byte rate_live_z_index = 6;
		private readonly byte rate_extra_z_index = 7;
		private readonly byte rate_dead_z_index = 8;
		private readonly byte rate_one_index = 9;
		private readonly byte rate_values_size = 10;
*/

extern "C" __global__  void cuda_update_live(int n_axons, float* tox, unsigned char* rate, float* detox, float* tox_prod, float on_death_tox, float k_detox_extra, float* death_tox_thres,
	unsigned int * axons_cent_pix, unsigned int* axons_inside_pix, unsigned int* axons_inside_pix_idx, unsigned int* axon_surr_rate, unsigned int* axon_surr_rate_idx,
	unsigned char* axon_state, unsigned char* axon_mask, int* num_alive_axons, int* death_itr, int iteration, int offset, int pixelNeighbourNumbers, float deathDetox, unsigned int* axon_death_timer, unsigned int iterToDeath)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	if (n < n_axons)
	{
		// [DWL]: I made death_tox_threshold to be an array INSTEAD of a constant 
		//			This way we can set the death threshold DIFFERENTLY for each axon
		//			The death is calculated at the head of the axon
		if (axon_state[n] == 2) {
			if (axon_death_timer[n] == 0) {

				for (int p = axons_inside_pix_idx[n]; p < axons_inside_pix_idx[n + 1]; p++)
				{
					int idx = axons_inside_pix[p];
					tox_prod[idx] = 0;
					detox[idx] = deathDetox;
					axon_mask[idx] = 3; // dead for display
				}
				axon_state[n] = 0;
			    death_itr[n] = iteration;
			    atomicAdd(&num_alive_axons[0], -1);
			}
			else {
				axon_death_timer[n]--;
			}
		}
		else if (axon_state[n] == 1 && tox[offset+axons_cent_pix[n]] >= death_tox_thres[n])
		{ 	// Kill the axon
			for (int p = axons_inside_pix_idx[n]; p < axons_inside_pix_idx[n + 1]; p++)
			{
				int idx = axons_inside_pix[p];

				
				//tox[offset+idx] += on_death_tox;
				if (tox_prod[idx] > 0) {
					tox_prod[idx] = on_death_tox;
				}
				//else {
					//detox[idx] = deathDetox;
				//}
				axon_mask[idx] = 2; // apoptotic for display
				
				/*
				int idxN = pixelNeighbourNumbers * idx;
				for (int i = 0; i < pixelNeigbourNumbers; i++) {
					rate[idxN + i] = diff_dead_index;
				}
				*/
				
			}

			for (int p = axon_surr_rate_idx[n], i = 0; p < axon_surr_rate_idx[n + 1]; p++, i++) {
				if (axon_surr_rate[p] != 0) {
					if ((i % pixelNeighbourNumbers) < 4)
						rate[axon_surr_rate[p]] = diff_dead_index;
					else
						rate[axon_surr_rate[p]] = diff_dead_z_index;
				}
			}
			
			//axon_state[n] = 0;
			//death_itr[n] = iteration;
			//atomicAdd(&num_alive_axons[0], -1);
			axon_state[n] = 2;
			axon_death_timer[n] = iterToDeath;
		}
	}
}

