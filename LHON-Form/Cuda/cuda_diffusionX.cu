
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_diffusionX(int* pix_idx, int pix_idx_num, unsigned short im_size,
	int tox_switch, float* tox, float* rate, float* detox, float* tox_prod, unsigned int* id_center_axon, 
	float on_death_tox, float k_rate_dead_axon, float k_detox_extra, float death_tox_thres,
	unsigned int* axons_cent_pix, unsigned int* axons_inside_pix, unsigned int* axons_inside_pix_idx, unsigned int* axon_surr_rate, unsigned int* axon_surr_rate_idx,
	bool* axon_is_alive, unsigned char* axon_mask, int* num_alive_axons, int* death_itr, int iteration)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		int xy0 = xy + im_size;
		int xy1 = xy - im_size;
		int xy2 = xy + 1;
		int xy3 = xy - 1;
		int xy4 = xy * 4;

		float *tox_new, *tox_old;

		if (tox_switch > 0) {
			tox_old = &tox[im_size*im_size];
			tox_new = &tox[0];
		}
		else {
			tox_new = &tox[im_size*im_size];
			tox_old = &tox[0];
		}

		float t = tox_old[xy];

		tox_new[xy] = t +
			(tox_old[xy0] - t) * rate[xy4] +
			(tox_old[xy1] - t) * rate[xy4 + 1] +
			(tox_old[xy2] - t) * rate[xy4 + 2] +
			(tox_old[xy3] - t) * rate[xy4 + 3] +
			tox_prod[xy];

		tox_new[xy] *= detox[xy];
		
		int n = id_center_axon[xy];
		if (n && axon_is_alive[n] && tox[axons_cent_pix[n]] > death_tox_thres)
		{ 	// Kill the axon
			for (int p = axons_inside_pix_idx[n]; p < axons_inside_pix_idx[n + 1]; p++)
			{
				int idx = axons_inside_pix[p];

				detox[idx] = k_detox_extra;
				tox[idx] += on_death_tox;
				tox_prod[idx] = 0;
				axon_mask[idx] = 2; // dead
			}

			for (int p = axon_surr_rate_idx[n]; p < axon_surr_rate_idx[n + 1]; p++)
				rate[axon_surr_rate[p]] = k_rate_dead_axon;

			/*
			int idx4 = 4 * idx;
			rate[idx4] = k_rate_dead_axon;
			rate[idx4 + 1] = k_rate_dead_axon;
			rate[idx4 + 2] = k_rate_dead_axon;
			rate[idx4 + 3] = k_rate_dead_axon;
			*/

			axon_is_alive[n] = false;
			death_itr[n] = iteration;
			atomicAdd(&num_alive_axons[0], -1);
		}
	}
	
}
