
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_adjustTOX(int* pix_idx, int pix_idx_num, float* tox, float toxMult)
{
	
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		tox[xy] =  tox[xy]*toxMult;
		
	}
}
