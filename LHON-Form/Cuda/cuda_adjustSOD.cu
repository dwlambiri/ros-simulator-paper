
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_adjustSOD(int* pix_idx, int pix_idx_num, float* detox, float detoxMult)
{
	
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		detox[xy] =  1-(1-detox[xy])*detoxMult;
		
	}
}
