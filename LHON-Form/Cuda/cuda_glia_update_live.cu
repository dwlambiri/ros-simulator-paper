
#include <hip/hip_runtime.h>

#define diff_membrane_index 2
#define diff_membrane_dead_index 3
#define diff_dead_index 4
#define diff_dead_z_index 8
#define diff_membrane_stress_index 9

#define GLIA_HEALTHY 1
#define GLIA_STRESS 2
#define GLIA_DEAD 0

#define GLIA_DISPLAY_HEALTHY 5
#define GLIA_DISPLAY_STRESS 6
#define GLIA_DISPLAY_DEAD 4

/*
* [DWL]
* Glia states:
* GLIA_HEALTHY --> GLIS_STRESS --> GLIA_DEAD
*/

/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
		private readonly byte rate_zero_index = 0;
        private readonly byte rate_live_index = 1;
        private readonly byte rate_membrane_index = 2;
        private readonly byte rate_membrane_dead_index = 3;
        private readonly byte rate_dead_index = 4;
        private readonly byte rate_extra_index = 5;
        private readonly byte rate_live_z_index = 6;
        private readonly byte rate_extra_z_index = 7;
        private readonly byte rate_dead_z_index = 8;
        private readonly byte rate_membrane_stress_index = 9;
        private readonly byte rate_one_index = 10;
        private readonly byte rate_values_size = 11;
*/


extern "C" __global__  void cuda_glia_update_live(int n_glias, float* tox, float* detox, float* tox_prod, float k_stress_glia_tox_prod, float k_healthy_glia_tox_prod, float h2s_tox_thres, float s2d_tox_thres, float s2h_tox_thres,
	unsigned int* glia_center, unsigned char* glia_state, unsigned int* glia_death_timer, unsigned char* simulation_mask, int offset, float k_death_glia_detox, unsigned int iterToDeath , int timer_reset)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	// [DWL] states: 0 == dead, 1== alive, 2 == stress

	if (n < n_glias)
	{
		// [DWL]: 
		int center = glia_center[n];
		if (glia_state[n] == GLIA_STRESS) {
			if (glia_death_timer[n] > iterToDeath || tox[offset + center] >=  s2d_tox_thres) {
				glia_state[n] = GLIA_DEAD; //dead
				tox_prod[center] = -1;
				detox[center] = k_death_glia_detox;
				simulation_mask[center] = GLIA_DISPLAY_DEAD; // for display
			}
			else if(tox[offset + center] < s2h_tox_thres) {
				glia_state[n] = GLIA_HEALTHY; // move back to healthy
				tox_prod[center] = k_healthy_glia_tox_prod;
				simulation_mask[center] = GLIA_DISPLAY_HEALTHY; // for display
				if(timer_reset) glia_death_timer[n] = 0;
			}
			else {
				glia_death_timer[n]++;
			}
		}
		else if (glia_state[n] == GLIA_HEALTHY )
		{ 	

			if (tox[offset + center] >= h2s_tox_thres) {
				tox_prod[center] = k_stress_glia_tox_prod;
				glia_state[n] = GLIA_STRESS;
				glia_death_timer[n]++;
				simulation_mask[center] = GLIA_DISPLAY_STRESS;  // for display
			}
			else {
				if (glia_death_timer[n] > 0) glia_death_timer[n]--;
			}
		}
	}
}

