
#include <hip/hip_runtime.h>
﻿
#define rate_extra_index 5
#define rate_extra_z_index 7

#define pixelInTissue 127
#define pixelOutsideOfTissue 255
#define noOpticZone  -1
#define  pi 3.141592654f

#define opticNerveZones 8
#define rate_tissue_index_xy 10
#define rate_tissue_index_z 11


/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
		private readonly byte rate_zero_index = 0;
        private readonly byte rate_live_index = 1;
        private readonly byte rate_membrane_index = 2;
        private readonly byte rate_membrane_dead_index = 3;
        private readonly byte rate_dead_index = 4;
        private readonly byte rate_extra_index = 5;
        private readonly byte rate_live_z_index = 6;
        private readonly byte rate_extra_z_index = 7;
        private readonly byte rate_dead_z_index = 8;
        private readonly byte rate_membrane_stress_index = 9;
        private readonly byte rate_tissue_index_xy = 10;
        private readonly byte rate_tissue_index_z = 11;
        private readonly byte rate_one_index = 12;
        private readonly byte rate_values_size = 13;
*/

extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_detox_extra,
	unsigned char* pix_out_of_nerve, unsigned char* rate, float* detox, unsigned int rate_dimensions, float* tox_prod, float outsideDetox, float* tox_dev, float initial_tox, unsigned char* simulation_point_mask, int* regionalMask, float* topoZoneAnglesRadian, float outsideToxProd, float noToxValue)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xyN = xy * rate_dimensions;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;
		
		if (!outside)
		{
			rate[xyN] = rate_extra_index;
			rate[xyN + 1] = rate_extra_index;
			rate[xyN + 2] = rate_extra_index;
			rate[xyN + 3] = rate_extra_index;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = rate_extra_z_index;
				rate[xyN + 5] = rate_extra_z_index;
			}
			

			detox[xy] = k_detox_extra;
			tox_prod[xy] = noToxValue;			
			tox_dev[xy] = initial_tox;
			simulation_point_mask[xy] = pixelInTissue;
 
			if (pix_out_of_nerve[xy + im_size]) { rate[xyN] = rate_tissue_index_xy; pix_out_of_nerve[xy + im_size] = 2; }
			if (pix_out_of_nerve[xy - im_size]) { rate[xyN + 1] = rate_tissue_index_xy; pix_out_of_nerve[xy - im_size] = 2; }
			if (pix_out_of_nerve[xy + 1]) { rate[xyN + 2] = rate_tissue_index_xy; pix_out_of_nerve[xy + 1] = 2; }
			if (pix_out_of_nerve[xy - 1]) { rate[xyN + 3] = rate_tissue_index_xy; pix_out_of_nerve[xy - 1] = 2; }
			float theta = 0;
            if(dx == 0)
            {
               if (dy >= 0)
                   theta = pi / 2;
               else
                   theta = pi * 3 / 4;

            }else
            {

               if(dx < 0)
               {
                   if (dy >= 0)
                      theta = atanf((float)(-1.0)*dy / dx);
                   else
                      theta = 2*pi - atanf((float)dy / dx);
                   }
                   else
                   {
                     if (dy >= 0)
                        theta = pi - atanf((float) dy / dx);
                     else
                        theta = pi + atanf((float)(-1.0)* dy/ dx);
                   }
           }

               

           for (int q = 0; q < opticNerveZones+1; q++)
           {
              if (theta < topoZoneAnglesRadian[q])
              {
                  regionalMask[xy] = (int)((q+ (opticNerveZones-1))% opticNerveZones);
                  break;
              }
           }
		}
		else {
			rate[xyN] = rate_tissue_index_xy;
			rate[xyN + 1] = rate_tissue_index_xy;
			rate[xyN + 2] = rate_tissue_index_xy;
			rate[xyN + 3] = rate_tissue_index_xy;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = rate_tissue_index_z;
				rate[xyN + 5] = rate_tissue_index_z;
			}
			detox[xy] = outsideDetox;
			tox_prod[xy] = outsideToxProd;
			tox_dev[xy] = 0;
			simulation_point_mask[xy] = pixelOutsideOfTissue;
			regionalMask[xy] = noOpticZone;
 		}
	}
}

//bool[,] pix_out_of_nerve = new bool[im_size, im_size];
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int dx = x - nerve_cent_pix;
//        int dy = y - nerve_cent_pix;
//        int dis2 = dx * dx + dy * dy;

//        bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
//        pix_out_of_nerve[x, y] = outside;
//        if (!outside)
//        {
//            pix_idx[pix_idx_num++] = x * im_size + y;
//            for (uint k = 0; k < 4; k++)
//                rate[x, y, k] = k_rate_extra;
//            detox[x, y] = k_detox_extra;
//        }
//    }

